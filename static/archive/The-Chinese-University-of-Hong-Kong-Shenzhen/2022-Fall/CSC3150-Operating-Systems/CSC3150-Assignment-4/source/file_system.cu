﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;
__device__ bool from_fs_write = false;
__device__ u32 invert_location[32768];

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  for(u32 addr = fs -> SUPERBLOCK_SIZE; addr < fs -> FILE_BASE_ADDRESS; addr += fs -> FCB_SIZE){
    for(u32 offset = 0; offset < fs -> MAX_FILENAME_SIZE; ++offset){
      fs -> volume[addr + offset] = '\0';
    }
  }
  for(u32 addr = 0; addr < fs -> SUPERBLOCK_SIZE; ++addr){
    fs -> volume[addr] = 0;
  }
  return;
}

__device__ u32 get_location(FileSystem *fs, u32 addr){
  return
    fs -> volume[addr + 20] * 256 +
    fs -> volume[addr + 21]; // FCB[20..21]
}

__device__ void set_location(FileSystem *fs, u32 addr, u32 value){
  invert_location[value] = addr;

  fs -> volume[addr + 20] = value / 256;
  fs -> volume[addr + 21] = value % 256; // FCB[20..21]
}

__device__ u32 get_size(FileSystem *fs, u32 addr){
  return
    fs -> volume[addr + 22] * 256 +
    fs -> volume[addr + 23]; // FCB[22..23]
}

__device__ void set_size(FileSystem *fs, u32 addr, u32 value){
  fs -> volume[addr + 22] = value / 256;
  fs -> volume[addr + 23] = value % 256; // FCB[22..23]
}

__device__ u32 get_create(FileSystem *fs, u32 addr){
  return
    fs -> volume[addr + 24] * 256 * 256 * 256 +
    fs -> volume[addr + 25] * 256 * 256 +
    fs -> volume[addr + 26] * 256 +
    fs -> volume[addr + 27]; // FCB[24..27]
}

__device__ void set_create(FileSystem *fs, u32 addr, u32 value){
  fs -> volume[addr + 24] = value / (256 * 256 * 256);
  value %= (256 * 256 * 256);

  fs -> volume[addr + 25] = value / (256 * 256);
  value %= (256 * 256);

  fs -> volume[addr + 26] = value / (256);
  value %= (256);

  fs -> volume[addr + 27] = value; // FCB[24..27]
  return;
}

__device__ u32 get_last_modified(FileSystem *fs, u32 addr){
  return
    fs -> volume[addr + 28] * 256 * 256 * 256 +
    fs -> volume[addr + 29] * 256 * 256 +
    fs -> volume[addr + 30] * 256 +
    fs -> volume[addr + 31]; // FCB[28..31]
}

__device__ void set_last_modified(FileSystem *fs, u32 addr, u32 value){
  fs -> volume[addr + 28] = value / (256 * 256 * 256);
  value %= (256 * 256 * 256);

  fs -> volume[addr + 29] = value / (256 * 256);
  value %= (256 * 256);

  fs -> volume[addr + 30] = value / (256);
  value %= (256);

  fs -> volume[addr + 31] = value; // FCB[28..31]
  return;
}

__device__ bool block_is_filled(FileSystem *fs, u32 block){
  return fs -> volume[block / 8] & (1 << (block % 8));
}

__device__ void block_toggle(FileSystem *fs, u32 block){
  fs -> volume[block / 8] ^= (1 << (block % 8));
  return;
}

__device__ u32 compact(FileSystem *fs){
  u32 last = -1;
  for(u32 block = 0; block < (1 << 15); ++block){
    if(!block_is_filled(fs, block) && last == -1){
      last = block;
    } else if(block_is_filled(fs, block) && last != -1){
      u32 addr = invert_location[block];
      u32 size = get_size(fs, addr);

      for(u32 i = 0; i < size / 32; ++i){
        block_toggle(fs, last + i);
        block_toggle(fs, block + i); // shift indicators to the left

        for(u32 j = 0; j < 32; ++j){
          fs -> volume[fs -> FILE_BASE_ADDRESS + (last + i) * 32 + j] = fs -> volume[fs -> FILE_BASE_ADDRESS + (block + i) * 32 + j];
          fs -> volume[fs -> FILE_BASE_ADDRESS + (block + i) * 32 + j] = 0;
        } // shift contents to the left
      } 

      set_location(fs, addr, last); // set location back

      last = -1;
    }
  }

  for(u32 block = 0; block < (1 << 15); ++block){
    if(!block_is_filled(fs, block)){
      return block;
    }
  }

  printf("[Error] no available blocks left.\n");
  
  return -1;
}

__device__ void block_check(FileSystem *fs, u32 le, u32 ri){
  for(u32 i = le; i < ri; ++i){
    if(block_is_filled(fs, i)){
      printf("1");
    } else {
      printf("0");
    }
  }
  printf("\n");
}

__device__ u32 strlen(char *s){
  int len = 0;
  while(s[len] != '\0'){
    len++;
  }

  return len;
}

__device__ void strcpy(char *s, char *t){
  for(u32 k = 0; k < strlen(t); ++k){
    s[k] = t[k];
  }

  return;
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op){
  /* Implement open operation here */
  gtime++;
	
  // Search FCB from volume[2^12 .. 2^12 + 2^15)
  for(u32 addr = fs -> SUPERBLOCK_SIZE; addr < fs -> FILE_BASE_ADDRESS; addr += fs -> FCB_SIZE){
    if(fs -> volume[addr] == '\0') continue;

    int k = 0;
    s += '\0';
    while(k < strlen(s) && k < fs -> MAX_FILENAME_SIZE){
      if(fs -> volume[addr + k] != s[k]) break;
      k++;
    } // FCB[0 .. 19]

    // Case for file exists
    if(k == strlen(s)){
      u32 start_block = get_location(fs, addr);

      // Case for file exists and operation mode is write
      if(op == G_WRITE && from_fs_write){
        for(u32 i = 0; i < get_size(fs, addr) / 32; ++i){
          block_toggle(fs, get_location(fs, addr) + i);
        } // set bitmap from 1 to 0 (assume the implementation is correct)
        
        set_size(fs, addr, 0); // set size to 0
        
        start_block = compact(fs);
        
        set_location(fs, addr, start_block); // set location to a new starting block

        set_last_modified(fs, addr, gtime); // set last modified time
      }

      return start_block;
    }
  }

  // Case for file does not exist and operation mode is read
  if(op == G_READ){
    printf("[Error] trying to read a file that does not exist.\n");

    return -1;
  }
  // Case for file does not exist and operation mode is write
  else if(op == G_WRITE){
    for(u32 addr = fs -> SUPERBLOCK_SIZE; addr < fs -> FILE_BASE_ADDRESS; addr += fs -> FCB_SIZE){
      if(fs -> volume[addr] == '\0'){
        s += '\0';
        for(int k = 0; k < strlen(s) && k < fs -> MAX_FILENAME_SIZE; ++k){
          fs -> volume[addr + k] = s[k];
        }

        // printf("%d is now filled by: ", (addr - fs -> SUPERBLOCK_SIZE) / fs -> FCB_SIZE);
        // for(int k = 0; k < strlen(s); ++k){
        //   printf("%c", fs -> volume[addr + k]);
        // }
        // printf("\n");

        set_size(fs, addr, 0); // set size to 0

        u32 start_block = compact(fs);

        set_location(fs, addr, start_block); // set location to a new starting block

        set_create(fs, addr, gtime);

        set_last_modified(fs, addr, gtime); // set last modified time

        return start_block;
      }
    } // search for an empty location;

    printf("[Error] 1024 files already created.\n");

    return -1;
  }
  // Case for unknown operation mode
  else {
    printf("[Error] operation mode is unknown.\n");

    return -1;
  }
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp){
	/* Implement read operation here */
  gtime++;

  if(fp == -1){
    printf("The current fs_read process is now stopped due to an error.\n");

    return;
  }

  for(u32 i = 0; i < size; ++i){
    if(!block_is_filled(fs, fp + (i / 32))) continue;
    output[i] = fs -> volume[fs -> FILE_BASE_ADDRESS + fp * 32 + i];
  } // read contents

  return;
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp){
	/* Implement write operation here */
  gtime++;

  if(fp == -1){
    printf("The current fs_write process is now stopped due to an error.\n");

    return;
  }

  u32 addr = invert_location[fp];
	char s[20];
  for(int k = 0; k < fs -> MAX_FILENAME_SIZE; ++k){
    s[k] = fs -> volume[addr + k];
  }
  
  from_fs_write = true;
  fp = fs_open(fs, s, G_WRITE); // call fs_open (in case fs_write is called without fs_open)
  from_fs_write = false;

  if(fp == -1){
    printf("The current fs_write process is now stopped due to an error.\n");

    return;
  }

  addr = invert_location[fp];
  set_size(fs, addr, size);

  // printf("Try to toggle [%u, %u) from 0 to 1\n", fp, fp + size / 32);
  for(u32 i = 0; i < size / 32; ++i){
    // printf("Toggling %u...\n", fp + i);
    block_toggle(fs, fp + i);
  } // set bitmap from 0 to 1 (assume the implementation is correct)

  for(u32 i = 0; i < size; ++i){
    fs -> volume[fs -> FILE_BASE_ADDRESS + fp * 32 + i] = input[i];
  } // write contents

  set_last_modified(fs, addr, gtime); // set last modified time

  return 0;
}

__device__ void fs_gsys(FileSystem *fs, int op){
	/* Implement LS_D and LS_S operation here */
  gtime++;
	
  u32 n = 0;
  u32 rank[1024];
  u32 size[1024];
  u32 create[1024];
  u32 last_modified[1024];

  for(u32 addr = fs -> SUPERBLOCK_SIZE; addr < fs -> FILE_BASE_ADDRESS; addr += fs -> FCB_SIZE){
    if(fs -> volume[addr] == '\0') continue;

    // printf("%d is found as ", (addr - fs -> SUPERBLOCK_SIZE) / fs -> FCB_SIZE);
    // for(u32 offset = 0; offset < fs -> MAX_FILENAME_SIZE; ++offset){
    //   if(offset == 20) break;
    //   printf("%c", fs -> volume[addr + offset]);
    // }
    // printf("\n");

    rank[n] = (addr - fs -> SUPERBLOCK_SIZE) / fs -> FCB_SIZE;
    size[n] = get_size(fs, addr);
    create[n] = get_create(fs, addr);
    last_modified[n] = get_last_modified(fs, addr);
    n++;
  }

  if(n == 0){
    printf("[Error] no file found.\n");

    return;
  }
  
  // Sort use bubble sort algorithm
  for(u32 i = 0; i < n; ++i){
    for(u32 j = i + 1; j < n; ++j){
      if( (op == LS_D && last_modified[i] < last_modified[j]) ||
          (op == LS_S && (size[i] < size[j] || (size[i] == size[j] && create[i] > create[j])))
        ){
        u32 rank_tmp;
        rank_tmp = rank[i];
        rank[i] = rank[j];
        rank[j] = rank_tmp;

        u32 size_tmp;
        size_tmp = size[i];
        size[i] = size[j];
        size[j] = size_tmp;

        u32 create_tmp;
        create_tmp = create[i];
        create[i] = create[j];
        create[j] = create_tmp;

        u32 last_modified_tmp;
        last_modified_tmp = last_modified[i];
        last_modified[i] = last_modified[j];
        last_modified[j] = last_modified_tmp;
      }
    }
  }

  if(op == LS_D){
    printf("===sort by modified time===\n");
  } else if(op == LS_S){
    printf("===sort by file size===\n");
  } else {
    printf("[Error] operation mode is unknown.\n");

    return;
  }

  for(u32 i = 0; i < n; ++i){

    u32 file_location = rank[i];
    
    // printf("%u. ", file_location);

    char s[20];
    for(u32 j = 0; j < fs -> MAX_FILENAME_SIZE; ++j){
      s[j] = '\0';
    }

    for(u32 j = 0; j < fs -> MAX_FILENAME_SIZE; ++j){
      char c = fs -> volume[fs -> SUPERBLOCK_SIZE + file_location * fs -> FCB_SIZE + j];
      if(c == '\0') break;
      s[j] = c;
      // printf("%c", c);
    }
    printf("%s", s);

    if(op == LS_S) printf(" %u", size[i]);

    printf("\n");
  }

  // printf("Total files: %u\n", n);

  return;
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s){
	/* Implement rm operation here */
  gtime++;

  if(op == RM){
    for(u32 addr = fs -> SUPERBLOCK_SIZE; addr < fs -> FILE_BASE_ADDRESS; addr += fs -> FCB_SIZE){
      if(fs -> volume[addr] == '\0') continue;

      s += '\0';

      int k = 0;
      while(k < strlen(s) && k < fs -> MAX_FILENAME_SIZE){
        if(fs -> volume[addr + k] != s[k]) break;
        k++;
      } // FCB[0 .. 19]

      if(k == strlen(s)){
        for(u32 offset = 0; offset < fs -> MAX_FILENAME_SIZE; ++offset){
          fs -> volume[addr + offset] = '\0';
        }

        for(u32 i = 0; i < get_size(fs, addr) / 32; ++i){
          block_toggle(fs, get_location(fs, addr) + i);
        } // set bitmap from 1 to 0 (assume the implementation is correct)

        return;
      }
    }

    printf("[Error] file does not exist.\n");
  } else {
    printf("[Error] operation mode is unknown.\n");
  }

  return;
}
