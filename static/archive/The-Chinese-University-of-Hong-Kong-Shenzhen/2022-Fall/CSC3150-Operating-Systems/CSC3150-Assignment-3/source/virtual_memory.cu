﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

const u32 MAX_FRAME_ADDRESS = 0x80000000;
const int MAX_BIT = 32;
const int MAX_PAGE_ENTRIES = 262144;

__device__ int new_page_table[MAX_PAGE_ENTRIES];

__device__ void init_invert_page_table(VirtualMemory *vm) {
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = MAX_FRAME_ADDRESS; // invalid := MSB is 1
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ int findFrame(VirtualMemory *vm, u32 frame_number){
  for(int page = 0; page < vm -> PAGE_ENTRIES; page = page + 1){
    if(vm -> invert_page_table[page + vm -> PAGE_ENTRIES] == frame_number){
      return page;
    }
  }

  return -1;
}

__device__ void leftRotate(VirtualMemory *vm, u32 frame){
  const int PAGE_ENTRIES = vm -> PAGE_ENTRIES;

  int tmp = vm -> invert_page_table[PAGE_ENTRIES + frame];
  for(int page = frame; page < PAGE_ENTRIES - 1; page = page + 1){
    vm -> invert_page_table[page + PAGE_ENTRIES] = vm -> invert_page_table[page + PAGE_ENTRIES + 1];
  }
  vm -> invert_page_table[2 * PAGE_ENTRIES - 1] = tmp;
  return;
  int page = frame;
  while(page + 1 < PAGE_ENTRIES){
    new_page_table[page + PAGE_ENTRIES] =
      vm -> invert_page_table[page + PAGE_ENTRIES + 1];
    page = page + 1;
  }
  new_page_table[2 * PAGE_ENTRIES - 1] = vm -> invert_page_table[frame + PAGE_ENTRIES];
  // Directly copy the values back to vm's invert_page_table
  while(frame < PAGE_ENTRIES){
    vm -> invert_page_table[frame + PAGE_ENTRIES] =
      new_page_table[frame + PAGE_ENTRIES];
    frame = frame + 1;
  }

  return;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  const int PAGE_ENTRIES = vm -> PAGE_ENTRIES;

  // printf("read: %d\n", addr);

  /* Complete vm_read function to read single element from data buffer */
  u32 page_number, page_offset, frame_number;
  bool fault; 
  int page, frame;

  // Decipher page_address to page_number and page_offset
  page_number = addr / MAX_BIT;
  page_offset = addr % MAX_BIT;
  frame_number = -1;

  // Inspect for page faults
  fault = true;
  page = 0;
  while(page < PAGE_ENTRIES){
    if(page_number == vm -> invert_page_table[page]){
      fault = false;
      // Save page in the frame_number
      frame_number = page;
      break;
    }

    page = page + 1;
  }

  // Fault case:
  if(fault){
    *vm -> pagefault_num_ptr += 1;
    frame_number = vm -> invert_page_table[PAGE_ENTRIES];

    // Move information to main memory
    for(int bit = 0; bit < MAX_BIT; ++bit){
        uchar &str = vm -> storage[(vm -> invert_page_table[frame_number]) * MAX_BIT + bit];
        uchar &buf = vm -> buffer[frame_number * MAX_BIT + bit];
        str = buf;
        buf = vm -> storage[page_number * MAX_BIT + bit];
    }

    vm -> invert_page_table[frame_number] = page_number;
  }
  
  // Find frame_number in the frame table
  page = frame = findFrame(vm, frame_number);

  // Perform a left rotation for invert_page_table[frame + PAGE_ENTRIES .. 2 PAGE_ENTRIES-1]
  leftRotate(vm, frame);

  // Copy the left-rotated values from vm's invert_page_table to new_page_table
  // while(page + 1 < PAGE_ENTRIES){
  //   new_page_table[page + PAGE_ENTRIES] =
  //     vm -> invert_page_table[page + PAGE_ENTRIES + 1];
  //   page = page + 1;
  // }
  // new_page_table[2 * PAGE_ENTRIES - 1] = vm -> invert_page_table[frame];
  // // Directly copy the values back to vm's invert_page_table
  // while(frame < PAGE_ENTRIES){
  //   vm -> invert_page_table[frame + PAGE_ENTRIES] =
  //     new_page_table[frame + PAGE_ENTRIES];

  //   frame = frame + 1;
  // }

  return vm -> buffer[frame_number * MAX_BIT + page_offset];
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  const int PAGE_ENTRIES = vm -> PAGE_ENTRIES;
  // printf("write: %c to %d\n", value, addr);

  u32 page_number, page_offset, frame_number;
  bool fault; 
  int page, frame;

  // Decipher page_address to page_number and page_offset
  page_number = addr / MAX_BIT;
  page_offset = addr % MAX_BIT;
  frame_number = -1;

  // Inspect for page faults
  fault = true;
  page = 0;
  while(page < PAGE_ENTRIES){
    if(page_number == vm -> invert_page_table[page]){
      fault = false;
      // Save page in the frame_number
      frame_number = page;
      break;
    }

    page = page + 1;
  }

  // if(addr < 32 * 1023 && addr % 1024 == 992){
  //   printf("%d's fault is %d\n", addr, fault);
  // }

  if(fault){
    *vm -> pagefault_num_ptr += 1;
    // if(addr < 32 * 1023 && addr % 1024 == 992){
    //   printf("counted, now page fault equals to %d\n", *vm -> pagefault_num_ptr);
    // }
    // if(*vm -> pagefault_num_ptr % 1024 == 0)
    //   printf("write: %c to %d\n", value, addr);
    frame_number = vm -> invert_page_table[PAGE_ENTRIES];

    // Send information from buffer to storage if frame is full
    if(vm -> invert_page_table[frame_number] != MAX_FRAME_ADDRESS){
      for(int bit = 0; bit < MAX_BIT; ++bit){
        // printf("Storage Size (%d), accessed: %d\n", vm -> STORAGE_SIZE,
        //   vm -> invert_page_table[frame_number] * MAX_BIT + bit);
        // printf("Buffer Size (%d), accessed: %d\n", vm -> PHYSICAL_MEM_SIZE,
        //   frame_number * MAX_BIT + bit);
        // if(vm -> STORAGE_SIZE <= vm -> invert_page_table[frame_number] * MAX_BIT + bit ||
        //   vm -> PHYSICAL_MEM_SIZE <= frame_number * MAX_BIT + bit){
        //     printf("SKIPPED!\n");
        //     continue;
        // }
        vm -> storage[vm -> invert_page_table[frame_number] * MAX_BIT + bit]
          = vm -> buffer[frame_number * MAX_BIT + bit];
      }
    }

    vm -> invert_page_table[frame_number] = page_number;
  }

  vm -> buffer[frame_number * MAX_BIT + page_offset] = value;

  // Find frame_number in the frame table
  page = frame = findFrame(vm, frame_number);

  // Perform a left rotation for invert_page_table[frame + PAGE_ENTRIES .. 2 PAGE_ENTRIES-1]
  leftRotate(vm, frame);

  // Copy the left-rotated values from vm's invert_page_table to new_page_table
  // while(page + 1 < PAGE_ENTRIES){
  //   new_page_table[page + PAGE_ENTRIES] =
  //     vm -> invert_page_table[page + PAGE_ENTRIES + 1];
  //   page = page + 1;
  // }
  // new_page_table[2 * PAGE_ENTRIES - 1] = vm -> invert_page_table[frame];
  // // Directly copy the values back to vm's invert_page_table
  // while(frame < PAGE_ENTRIES){
  //   vm -> invert_page_table[frame + PAGE_ENTRIES] =
  //     new_page_table[frame + PAGE_ENTRIES];

  //   frame = frame + 1;
  // }

  // if(addr == 131071){
  //   for(int i = 0; i < PAGE_ENTRIES; ++i){
  //     if(i != 0) printf(",");
  //     printf("%d", vm -> invert_page_table[i]);
  //   }
  //   printf("\n");
  // }

  return;
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  const int PAGE_ENTRIES = vm -> PAGE_ENTRIES;

  // printf("after write, the number of page fault is: %d\n", *vm -> pagefault_num_ptr);
  // printf("snapshot: %d\n", input_size);

  /* Complete snapshot function together with vm_read to load elements from data
   * to result buffer */
  int page_address = 0;
  while(page_address < input_size){
    u32 actual_page_number, page_number, page_offset, frame_number;
    bool fault; 
    int page, frame;

    // Decipher page_address to page_number and page_offset
    actual_page_number = page_address / MAX_BIT;
    page_number = (page_address + offset) / MAX_BIT;
    page_offset = (page_address + offset) % MAX_BIT;
    frame_number = -1;

    // Inspect for page faults
    fault = true;
    page = 0;
    while(page < PAGE_ENTRIES){
      if(page_number == vm -> invert_page_table[page]){
        fault = false;
        // Save page in the frame_number
        frame_number = page;
        break;
      }

      page = page + 1;
    }

    // Fault case:
    if(fault){
      *vm -> pagefault_num_ptr += 1;
      frame_number = vm -> invert_page_table[PAGE_ENTRIES];

      // Move information to main memory
      for(int bit = 0; bit < MAX_BIT; ++bit){
        uchar &str = vm -> storage[(vm -> invert_page_table[frame_number]) * MAX_BIT + bit];
        uchar &buf = vm -> buffer[frame_number * MAX_BIT + bit];
        str = buf;
        buf = vm -> storage[page_number * MAX_BIT + bit];
      }

      vm -> invert_page_table[frame_number] = page_number;
    }
    
    // Print results from buffer
    for(int bit = 0; bit < MAX_BIT; ++bit){
      results[actual_page_number * MAX_BIT + bit] =
        (uchar) vm -> buffer[frame_number * MAX_BIT + bit];
      // results[page_number * MAX_BIT + bit] = 'W';
    }

    // Find frame_number in the frame table
    page = frame = findFrame(vm, frame_number);

    // Perform a left rotation for invert_page_table[frame + PAGE_ENTRIES .. 2 PAGE_ENTRIES-1]
    leftRotate(vm, frame);

    // Copy the left-rotated values from vm's invert_page_table to new_page_table
    // while(page + 1 < PAGE_ENTRIES){
    //   new_page_table[page + PAGE_ENTRIES] =
    //     vm -> invert_page_table[page + PAGE_ENTRIES + 1];
    //   page = page + 1;
    // }
    // new_page_table[2 * PAGE_ENTRIES - 1] = vm -> invert_page_table[frame];
    // // Directly copy the values back to vm's invert_page_table
    // while(frame < PAGE_ENTRIES){
    //   vm -> invert_page_table[frame + PAGE_ENTRIES] =
    //     new_page_table[frame + PAGE_ENTRIES];

    //   frame = frame + 1;
    // }

    page_address = page_address + 1;
  }
}

