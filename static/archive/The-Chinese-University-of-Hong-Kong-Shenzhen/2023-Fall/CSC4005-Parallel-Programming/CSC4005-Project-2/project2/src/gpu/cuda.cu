#include "hip/hip_runtime.h"
//
// Created by Yohandi on 2023/10/27.
// Email: yohandi@link.cuhk.edu.cn
//
// GPU Matrix Multiplication
//

#include "../matrix.hpp"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void matrixMultiply(int *matrix1, int *matrix2, int *result,
                               size_t M, size_t N, size_t K) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    int sum = 0;
    for (size_t k = 0; k < K; k++) {
      sum += matrix1[row * K + k] * matrix2[k * N + col];
    }

    result[row * N + col] = sum;
  }
}

int main(int argc, char **argv) {
  const std::string matrix1_path = argv[1];
  const std::string matrix2_path = argv[2];
  const std::string result_path = argv[3];

  Matrix matrix1 = Matrix::loadFromFile(matrix1_path);
  Matrix matrix2 = Matrix::loadFromFile(matrix2_path);

  size_t M = matrix1.getRows();
  size_t N = matrix2.getCols();
  size_t K = matrix1.getCols();

  int *d_matrix1, *d_matrix2;
  int *d_result;

  hipMalloc((void **)&d_matrix1, M * K * sizeof(int));
  hipMalloc((void **)&d_matrix2, K * N * sizeof(int));
  hipMalloc((void **)&d_result, M * N * sizeof(int));

  int *flat_matrix1, *flat_matrix2;
  hipMallocManaged(&flat_matrix1, M * K * sizeof(int));
  hipMallocManaged(&flat_matrix2, K * N * sizeof(int));

  for (int i = 0; i < M; i++)
    for (int j = 0; j < K; j++)
      flat_matrix1[i * K + j] = matrix1[i][j];

  for (int i = 0; i < K; i++)
    for (int j = 0; j < N; j++)
      flat_matrix2[i * N + j] = matrix2[i][j];

  hipMemcpy(d_matrix1, flat_matrix1, M * K * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_matrix2, flat_matrix2, K * N * sizeof(int),
             hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float gpuDuration;

  dim3 threadsPerBlock(32, 32);
  dim3 blocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
              (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  matrixMultiply<<<blocks, threadsPerBlock>>>(d_matrix1, d_matrix2, d_result, M,
                                              N, K);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpuDuration, start, stop);

  int *flat_result = (int *)malloc(M * N * sizeof(int));
  if (flat_result == NULL) {
    std::cerr << "Memory allocation failed!" << std::endl;
    exit(1);
  }
  hipMemcpy(flat_result, d_result, M * N * sizeof(int),
             hipMemcpyDeviceToHost);

  Matrix result(M, N);
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      result[i][j] = flat_result[i * N + j];
    }
  }

  result.saveToFile(result_path);

  std::cout << "Output file to: " << result_path << std::endl;

  std::cout << "Multiplication Complete!" << std::endl;
  std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds"
            << std::endl;

  hipFree(flat_matrix1);
  hipFree(flat_matrix2);
  delete[] flat_result;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
