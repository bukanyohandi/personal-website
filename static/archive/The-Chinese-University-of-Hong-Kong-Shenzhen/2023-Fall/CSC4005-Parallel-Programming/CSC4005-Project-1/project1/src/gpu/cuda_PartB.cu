#include "hip/hip_runtime.h"
/*
    Created by Yohandi on 2023/10/5.
    Email: yohandi@link.cuhk.edu.cn

    CUDA implementation of applying a 3 x 3 filter to a JPEG image

    Few parts were taken from cuda_PartA.cpp created by Yang Yufan as reference

    GPU execution time: 23 ms
*/

#include <iostream>

#include <hip/hip_runtime.h> // CUDA Header

#include "utils.hpp"

const int FILTER_SIZE = 3;
__constant__ float filter[FILTER_SIZE][FILTER_SIZE] = {
    {1.0f / 9, 1.0f / 9, 1.0f / 9},
    {1.0f / 9, 1.0f / 9, 1.0f / 9},
    {1.0f / 9, 1.0f / 9, 1.0f / 9}};

struct Pixel {
  unsigned char r, g, b;
};

// CUDA kernel functon：filter RGB
__global__ void filterRGB(unsigned char *input, unsigned char *output,
                          int width, int height, int num_channels) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < width || idx >= width * (height - 1) || idx % width == 0 || idx % width == height - 1) // border
    return;

  if (idx < width * height) {
    Pixel *base_ptr = reinterpret_cast<Pixel *>(input) + idx;
    int sum_r = 0, sum_g = 0, sum_b = 0;

    sum_r += (base_ptr[-width - 1].r * filter[0][0]) +
             (base_ptr[-width].r * filter[0][1]) +
             (base_ptr[-width + 1].r * filter[0][2]) +
             (base_ptr[-1].r * filter[1][0]) + (base_ptr[0].r * filter[1][1]) +
             (base_ptr[1].r * filter[1][2]) +
             (base_ptr[width - 1].r * filter[2][0]) +
             (base_ptr[width].r * filter[2][1]) +
             (base_ptr[width + 1].r * filter[2][2]);

    sum_g += (base_ptr[-width - 1].g * filter[0][0]) +
             (base_ptr[-width].g * filter[0][1]) +
             (base_ptr[-width + 1].g * filter[0][2]) +
             (base_ptr[-1].g * filter[1][0]) + (base_ptr[0].g * filter[1][1]) +
             (base_ptr[1].g * filter[1][2]) +
             (base_ptr[width - 1].g * filter[2][0]) +
             (base_ptr[width].g * filter[2][1]) +
             (base_ptr[width + 1].g * filter[2][2]);

    sum_b += (base_ptr[-width - 1].b * filter[0][0]) +
             (base_ptr[-width].b * filter[0][1]) +
             (base_ptr[-width + 1].b * filter[0][2]) +
             (base_ptr[-1].b * filter[1][0]) + (base_ptr[0].b * filter[1][1]) +
             (base_ptr[1].b * filter[1][2]) +
             (base_ptr[width - 1].b * filter[2][0]) +
             (base_ptr[width].b * filter[2][1]) +
             (base_ptr[width + 1].b * filter[2][2]);

    Pixel *output_pixels = reinterpret_cast<Pixel *>(output);
    Pixel &out_pixel = output_pixels[idx];

    out_pixel.r = static_cast<unsigned char>(sum_r);
    out_pixel.g = static_cast<unsigned char>(sum_g);
    out_pixel.b = static_cast<unsigned char>(sum_b);
  }
}

int main(int argc, char **argv) {
  // Verify input argument format
  if (argc != 3) {
    std::cerr << "Invalid argument, should be: ./executable "
                 "/path/to/input/jpeg /path/to/output/jpeg\n";
    return -1;
  }
  // Read from input JPEG
  const char *input_filepath = argv[1];
  std::cout << "Input file from: " << input_filepath << "\n";
  auto input_jpeg = read_from_jpeg(input_filepath);
  // Allocate memory on host (CPU)
  auto filteredImage = new unsigned char[input_jpeg.width * input_jpeg.height *
                                         input_jpeg.num_channels];
  // Allocate memory on device (GPU)
  unsigned char *d_input;
  unsigned char *d_output;
  hipMalloc((void **)&d_input, input_jpeg.width * input_jpeg.height *
                                    input_jpeg.num_channels *
                                    sizeof(unsigned char));
  hipMalloc((void **)&d_output, input_jpeg.width * input_jpeg.height *
                                     input_jpeg.num_channels *
                                     sizeof(unsigned char));
  // Copy input data from host to device
  hipMemcpy(d_input, input_jpeg.buffer,
             input_jpeg.width * input_jpeg.height * input_jpeg.num_channels *
                 sizeof(unsigned char),
             hipMemcpyHostToDevice);

  // Apply filter to RGB
  hipEvent_t start, stop;
  float gpuDuration;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int blockSize = 128; // 256
  int numBlocks =
      (input_jpeg.width * input_jpeg.height * input_jpeg.num_channels +
       blockSize - 1) /
      blockSize;
  // int numBlocks = (input_jpeg.width * input_jpeg.height *
  // input_jpeg.num_channels) / blockSize + 1;
  hipEventRecord(start, 0); // GPU start time
  filterRGB<<<numBlocks, blockSize>>>(d_input, d_output, input_jpeg.width,
                                      input_jpeg.height,
                                      input_jpeg.num_channels);
  hipEventRecord(stop, 0); // GPU end time
  hipEventSynchronize(stop);
  // Print the result of the GPU computation
  hipEventElapsedTime(&gpuDuration, start, stop);
  // Copy output data from device to host
  hipMemcpy(filteredImage, d_output,
             input_jpeg.width * input_jpeg.height * input_jpeg.num_channels,
             hipMemcpyDeviceToHost);
  // Write filteredImage to output JPEG
  const char *output_filepath = argv[2];
  std::cout << "Output file to: " << output_filepath << "\n";
  JPEGMeta output_jpeg{filteredImage, input_jpeg.width, input_jpeg.height,
                       input_jpeg.num_channels, input_jpeg.color_space};
  if (write_to_jpeg(output_jpeg, output_filepath)) {
    std::cerr << "Failed to write output JPEG\n";
    return -1;
  }
  // Release allocated memory on device and host
  hipFree(d_input);
  hipFree(d_output);
  delete[] input_jpeg.buffer;
  delete[] filteredImage;
  std::cout << "Transformation Complete!" << std::endl;
  std::cout << "GPU Execution Time: " << gpuDuration << " milliseconds"
            << std::endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return 0;
}